#include "hip/hip_runtime.h"
/*
 * Implementation of Incomplete-LU preconditioned conjugate-gradient for symmetric PSD systems using CUDA 12.0.
 * The Matrix Vector multiplication has been implemented in CSR format with one warp taking care of one row of matrix
 * The sparse lower and upper triangular solver  have been implemented using cuSPARSE library.
 * 
 * Considering the ILU factorization is deprecated in the new version of cuSPARSE, I am calculating it manually on the host side.
 * 
 * Author: Harshal Raut
 */
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <iostream>
#include <time.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>	//cusparse library
#include <hipblas.h>	//cublas library
#include <sys/time.h>

extern "C" {
#include "helper.h"
#include "sequential.h"
}

// vecVec
#define BLOCK_DIM_VEC 32

//matVec
#define NB_ELEM_MAT 32
#define BLOCK_SIZE_MAT 32

/*
 * --Naive implementation--
 * Computes a (square) matrix vector product
 * Input: pointer to 1D-array-stored matrix, 1D-array-stored vector
 * Stores the product in memory at the location of the pointer out
 */
__global__ void matVec(float* A, float* b, float* out, int* SIZE) {
	unsigned int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	if (index_x < *SIZE) {
		float tmp = 0;
		for (int i = 0; i < *SIZE; i++) {
			tmp += b[i] * A[*SIZE * index_x + i];
		}
		out[index_x] = tmp;
	}
}

/*
 * --More efficient implementation--
 * Computes a (square) symmetric matrix vector product
 * Input: pointer to 1D-array-stored matrix, 1D-array-stored vector
 * Stores the product in memory at the location of the pointer out
 */
__global__ void matVec2(float* ACSR, int* IA, int* JA, int* SIZE, float* b, float* out) {


	__shared__ volatile float t_sum[BLOCK_SIZE_MAT]; // thread sum
	int j;

	int t_id =  blockDim.x * blockIdx.x + threadIdx.x; // thread id
	int warpsPerBlock = blockDim.x / 32 ;
	int row = threadIdx.x / 32 + blockIdx.x* warpsPerBlock; // one warp per row
	int t_warp = threadIdx.x & (32-1); // thread number within a given warp

	// don't compute for a row value greater than the total in our matrix!
	if (row < *SIZE){

		// compute running sum per thread in warp
		float dotProduct =0;
		// t_sum[threadIdx.x] = 0;

		for (j = IA[row] + t_warp; j < IA[row+1]; j += 32)
		{
			dotProduct += ACSR[j] * b[JA[j]];
		}

		t_sum[threadIdx.x] = dotProduct;

		__syncthreads();
		// Parallel reduction of result in shared memory for one warp
		if (t_warp < 16) t_sum[threadIdx.x] += t_sum[threadIdx.x+16];
		if (t_warp < 8) t_sum[threadIdx.x] += t_sum[threadIdx.x+8];
		if (t_warp < 4) t_sum[threadIdx.x] += t_sum[threadIdx.x+4];
		if (t_warp < 2) t_sum[threadIdx.x] += t_sum[threadIdx.x+2];
		if (t_warp < 1) t_sum[threadIdx.x] += t_sum[threadIdx.x+1];
		// first thread within warp contains desired y[row] result so write it to y
		if (t_warp == 0)
			out[row] = t_sum[threadIdx.x];
	}
	//__syncthreads();
}

/*
 * Computes the sum of 2 vectors
 * Input: pointer to 1D-array-stored vector, pointer to 1D-array-stored vector
 * Stores the sum in memory at the location of the pointer out
 */
__global__ void vecPlusVec(float* a, float* b, float* out, int* SIZE) {
	unsigned int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	if (index_x < *SIZE) {
		out[index_x] = b[index_x] + a[index_x];
	}
}

/*
 * Computes the sum of 2 vectors
 * Input: pointer to 1D-array-stored vector, pointer to 1D-array-stored vector
 * Stores the sum in memory at the location of the pointer out
 * Also 0's the vector b
 */
__global__ void vecPlusVec2(float* a, float* b, float* out, int* SIZE) {
	unsigned int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	if (index_x < *SIZE) {
		out[index_x] = b[index_x] + a[index_x];
		b[index_x] = 0.0;
	}
}

/*
 * Computes the difference of 2 vectors
 * Input: pointer to 1D-array-stored vector, pointer to 1D-array-stored vector
 * Stores the sum in memory at the location of the pointer out
 */
__global__ void vecMinVec(float* a, float* b, float* out, int* SIZE) {
	unsigned int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	if (index_x < *SIZE) {
		out[index_x] = a[index_x] - b[index_x];
	}
}

/*
 * --Naive implementation--
 * Computes the inner product of 2 vectors
 * Input: pointer to 1D-array-stored vector, pointer to 1D-array-stored vector
 * Stores the product in memory at the location of the pointer out
 */
__global__ void vecVec(float* a, float* b, float* out, int* SIZE) {
	unsigned int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	float tmp = 0.0;
	if (index_x == 0) {
		for (int i = 0; i < *SIZE; i++) {
			tmp += b[i] * a[i];
		}
		*out = tmp;
	}
}

/*
 * --More efficient implementation--
 * Computes the inner product of 2 vectors
 * Input: pointer to 1D-array-stored vector, pointer to 1D-array-stored vector
 * Stores the product in memory at the location of the pointer out
 */
__global__ void vecVec2(float* a, float* b, float* out, int* SIZE) {
	// each block has it's own shared_tmp of size BLOCK_DIM_VEC
	__shared__ volatile float shared_tmp[BLOCK_DIM_VEC];

	// needed for atomicAdd
	if (threadIdx.x + blockDim.x * blockIdx.x == 0) {
		*out = 0.0;
	}


	if (blockIdx.x * blockDim.x + threadIdx.x < *SIZE) {
		shared_tmp[threadIdx.x] = a[blockIdx.x * blockDim.x + threadIdx.x]
				* b[blockIdx.x * blockDim.x + threadIdx.x];
	} else {
		// needed for the reduction
		shared_tmp[threadIdx.x] = 0.0;
	}

	// reduction within block
	for (int i = blockDim.x / 2; i >= 1; i = i / 2) {
		// threads access memory position written by other threads so sync is needed
		__syncthreads();
		if (threadIdx.x < i) {
			shared_tmp[threadIdx.x] += shared_tmp[threadIdx.x + i];
		}
	}
	__syncthreads();
	// atomic add the partial reduction in out
	if (threadIdx.x == 0) {
		atomicAdd(out, shared_tmp[0]);
	}
}

/*
 * Computes the product of a scalar with a vector
 * Input: pointer to scalar, pointer to 1D-array-stored vector
 * Stores the sum in memory at the location of the pointer out
 */
__global__ void scalarVec(float* scalar, float* a, float* out, int* SIZE) {
	unsigned int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	if (index_x < *SIZE) {
		out[index_x] = a[index_x] * *scalar;
	}
}

/*
 * Copies the content of vector in to vector out
 * Input: pointer to 1D-array-stored vector, pointer to 1D-array-stored vector
 */
__global__ void memCopy(float* in, float* out, int* SIZE) {
	unsigned int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	if (index_x < *SIZE) {
		out[index_x] = in[index_x];
	}
}

/*
 * Computes the quotient of 2 scalars
 * Input: pointer to scalar, pointer to scalar
 * Stores the quotient in memory at the location of the pointer out
 */
__global__ void divide(float* num, float* den, float* out) {
	unsigned int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	if (index_x == 0) {
		*out = *num / *den;
	}
}

/*
 * Main CG solver
 * All the given pointers are device pointers, with correct initial values
 */

void solveCG_cuda(float* d_A, float* A_CSR, int* JA, int* IA, float* d_UD, int* d_JU, int* d_IU, float* d_LD, int* d_JL, int* d_IL, float* b, float* x, float* p, float* r, float* z, float* temp,
		float* alpha, float* beta, float* r_norm, float* r_norm_old,
		float* temp_scal, float* h_x, float* h_r_norm, int M, int NNZ, int NNL, int NNU) {

	dim3 vec_block_dim(BLOCK_DIM_VEC); 
	dim3 vec_grid_dim((M + BLOCK_DIM_VEC - 1) / BLOCK_DIM_VEC); 
	dim3 mat_block_dim(BLOCK_SIZE_MAT);
	dim3 mat_grid_dim((M+BLOCK_SIZE_MAT - 1)  / BLOCK_SIZE_MAT * 32); //because each warp is taking care of one row of matrix during Mat-Vec multiplication

	float* numerator;
	hipMalloc((void **) &numerator, sizeof(float));

	float* denominator;
	hipMalloc((void **) &denominator, sizeof(float));

	int* SIZE;
	hipMalloc((void**)&SIZE, sizeof(int));
	hipMemcpy(SIZE, &M, sizeof(int), hipMemcpyHostToDevice);

    hipsparseHandle_t handle;
    hipsparseCreate(&handle);

	hipsparseSpMatDescr_t matL, matU;
	hipsparseDnVecDescr_t vecB, vecX, vecZ;
	hipsparseCreateCsr(&matL, M, M, NNL,
					d_IL, d_JL, d_LD,
					HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
					HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
	hipsparseCreateCsr(&matU, M, M, NNU,
					d_IU, d_JU, d_UD,
					HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
					HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
	hipsparseCreateDnVec(&vecB, M, r, HIP_R_32F);
	hipsparseCreateDnVec(&vecX, M, temp, HIP_R_32F);
	hipsparseCreateDnVec(&vecZ, M, z, HIP_R_32F);

	hipsparseSpSVDescr_t spsvDescrL;
	hipsparseSpSV_createDescr(&spsvDescrL);

	hipsparseSpSVDescr_t spsvDescrU;
	hipsparseSpSV_createDescr(&spsvDescrU);

	size_t bufferSize = 0;
	size_t stmp = 0;
	float alpha2 = 1.0f;
	hipsparseSpSV_bufferSize(handle, HIPSPARSE_OPERATION_TRANSPOSE,
							&alpha2, matU, vecB, vecX, HIP_R_32F,
							HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, &stmp);
	if (stmp > bufferSize)	bufferSize = stmp;

	hipsparseSpSV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
							&alpha2, matU, vecX, vecZ, HIP_R_32F,
							HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU, &stmp);
	if (stmp > bufferSize)	bufferSize = stmp;
	void* dBuffer;
	hipMalloc(&dBuffer, bufferSize);

	// Analysis step
	hipsparseSpSV_analysis(handle, HIPSPARSE_OPERATION_TRANSPOSE,
						&alpha2, matU, vecB, vecX, HIP_R_32F,
						HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, dBuffer);
	hipsparseSpSV_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
						&alpha2, matU, vecX, vecZ, HIP_R_32F,
						HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU, dBuffer);

	vecVec2<<<vec_grid_dim, vec_block_dim>>>(r, r, r_norm_old, SIZE);

	int k = 0;
	long micro_begin_gpu = getMicrotime();
	while ((k < MAX_ITER) && (*h_r_norm > EPS)) {


        // preconditioner application: z = U^-1 L^-1 r
        hipsparseSpSV_solve(handle, HIPSPARSE_OPERATION_TRANSPOSE,
						&alpha2, matU, vecB, vecX, HIP_R_32F,
						HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL);
        hipsparseSpSV_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
						&alpha2, matU, vecX, vecZ, HIP_R_32F,
						HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU);

        if (k == 0)
        {
			memCopy<<<vec_grid_dim, vec_block_dim>>>(z, p, SIZE);
			vecVec2<<<vec_grid_dim, vec_block_dim>>>(r, z, numerator, SIZE);
        }
        else
        {
			// beta_k = ...
			memCopy<<<1, 1>>>(numerator, denominator, SIZE);
			vecVec2<<<vec_grid_dim, vec_block_dim>>>(r, z, numerator, SIZE);
			divide<<<1, 1>>>(numerator, denominator, beta);

			// p_{k+1} = ...
			scalarVec<<<vec_grid_dim, vec_block_dim>>>(beta, p, temp, SIZE);
			vecPlusVec2<<<vec_grid_dim, vec_block_dim>>>(z, temp, p, SIZE);
        }

		matVec2<<< mat_grid_dim, mat_block_dim>>>(A_CSR, IA, JA, SIZE, p, temp);

		// alpha_k = ...
		vecVec2<<<vec_grid_dim, vec_block_dim>>>(p, temp, temp_scal, SIZE); //temp_scalar is denominator for alpha
		divide<<<1, 1>>>(numerator, temp_scal, alpha);

		// r_{k+1} = ...
		scalarVec<<<vec_grid_dim, vec_block_dim>>>(alpha, temp, temp, SIZE);
		vecMinVec<<<vec_grid_dim, vec_block_dim>>>(r, temp, r, SIZE);

		// x_{k+1} = ...
		scalarVec<<<vec_grid_dim, vec_block_dim>>>(alpha, p, temp, SIZE);
		vecPlusVec<<<vec_grid_dim, vec_block_dim>>>(x, temp, x, SIZE);

		vecVec2<<<vec_grid_dim, vec_block_dim>>>(r, r, r_norm, SIZE);

		// set r_norm_old to r_norm
		memCopy<<<1, 1>>>(r_norm, r_norm_old, SIZE);

		// copy to r_norm to CPU (to evaluate stop condition)
		hipMemcpy(h_r_norm, r_norm, sizeof(float), hipMemcpyDeviceToHost);
		k++;
		// printing the error and iteration number in CPU
		if(k % 1000==0) printf("iteraction:%d, Errnorm: %f \n", k, *h_r_norm);

	}
	long micro_end_gpu = getMicrotime();
	printf("iterations: %d \n",k);
	printf("norm: %f\n", *h_r_norm);
	printf("Time spent gpu per iter [s]: %e\n", (float) ((micro_end_gpu - micro_begin_gpu)/k) / 1e6);

	hipFree(dBuffer);
	hipsparseDestroy(handle);
    hipsparseSpSV_destroyDescr(spsvDescrU);
    hipsparseSpSV_destroyDescr(spsvDescrL);
    hipsparseDestroySpMat(matL);
    hipsparseDestroyDnVec(vecB);
    hipsparseDestroyDnVec(vecX);
    hipsparseDestroyDnVec(vecZ);

}



////////////////////////////////////////////////////////////////////////////////
// MAIN
////////////////////////////////////////////////////////////////////////////////
int main() {

	int M =32; // size of matrix

	// Parameters for sparce matrix
	double p_diag = 1.0; //probability of element along diagonal
	double p_nondiag = 0.15;

	// Parameters for CSR format for matrix A
	int *h_IA, *h_JA; 
	float *h_ACSR;
	int NNZ; // total non-zero elements

	//Parameters for CSR format for L and U from ilu decomposition
	int *h_IU, *h_IL, *h_JU, *h_JL;
	float *h_UCSR, *h_LCSR;
	int NNL, NNU;

	// allocate host memory
	float* h_A = generateA(M, p_diag, p_nondiag);
	float*h_LD, *h_UD;
	get_ILU_decomposition(&h_A, M, &h_LD, &h_UD);

	get_CSR_format(&h_A, &h_ACSR, &h_IA, &h_JA, &NNZ, M, p_diag, p_nondiag);

	//Get CSR for L and U matrix as well
	get_CSR(&h_LD, &h_LCSR, &h_IL, &h_JL, &NNL, M, NNZ);
	get_CSR(&h_UD, &h_UCSR, &h_IU, &h_JU, &NNU, M, NNZ);

	free(h_LD);
	free(h_UD);
	float* h_b = generateb(M);
	float* h_x = (float *) calloc(M, sizeof(float));
	float* h_r_norm = (float *) malloc(sizeof(float));
	*h_r_norm = 1.0;

	// allocate device memory
	float *d_A;
	float* d_ACSR, *d_LD, *d_UD;
	int* d_IA, *d_JA, *d_IL, *d_JL, *d_IU, *d_JU;
	float* d_b;
	float* d_x;
	float* d_p;
	float* d_r;
	float* d_z;
	float* d_temp;

	std::cout << "SIZE: " <<M << std::endl; //HR
	hipMalloc((void **) &d_A, M * M * sizeof(float));
	hipMalloc((void **) &d_ACSR, NNZ * sizeof(float));
	hipMalloc((void **) &d_JA, NNZ * sizeof(int));
	hipMalloc((void **) &d_IA, (M+1) * sizeof(int));
	hipMalloc((void **) &d_LD, NNL * sizeof(float));
	hipMalloc((void **) &d_JL, NNL * sizeof(int));
	hipMalloc((void **) &d_IL, (M+1) * sizeof(int));
	hipMalloc((void **) &d_UD, NNU * sizeof(float));
	hipMalloc((void **) &d_JU, NNU * sizeof(int));
	hipMalloc((void **) &d_IU, (M+1) * sizeof(int));

	hipMalloc((void **) &d_b, M * sizeof(float));
	hipMalloc((void **) &d_x, M * sizeof(float));
	hipMalloc((void **) &d_p, M * sizeof(float));
	hipMalloc((void **) &d_r, M * sizeof(float));
	hipMalloc((void **) &d_z, M * sizeof(float));
	hipMalloc((void **) &d_temp, M * sizeof(float));

	// copy host memory to device
	hipMemcpy(d_A, h_A, M * M * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_ACSR, h_ACSR, NNZ * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_JA, h_JA, NNZ * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_IA, h_IA, (M+1)* sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_UD, h_UCSR, NNU * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_JU, h_JU, NNU * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_IU, h_IU, (M+1)* sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_LD, h_LCSR, NNL * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_JL, h_JL, NNL * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_IL, h_IL, (M+1)* sizeof(int), hipMemcpyHostToDevice);


	hipMemcpy(d_b, h_b, M * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_x, h_x, M * sizeof(float), hipMemcpyHostToDevice);
	// assume x0 = 0
	hipMemcpy(d_p, h_b, M * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_r, h_b, M * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_z, h_b, M * sizeof(float), hipMemcpyHostToDevice);

	// 5 floats needed
	float* d_beta;
	float* d_alpha;
	float* d_r_norm;
	float* d_r_norm_old;
	float* d_temp_scal;
	hipMalloc((void **) &d_beta, sizeof(float));
	hipMalloc((void **) &d_alpha, sizeof(float));
	hipMalloc((void **) &d_r_norm, sizeof(float));
	hipMalloc((void **) &d_r_norm_old, sizeof(float));
	hipMalloc((void **) &d_temp_scal, sizeof(float));

	// run the main function
	solveCG_cuda(d_A,d_ACSR, d_JA, d_IA, d_UD, d_JU, d_IU, d_LD,d_JL, d_IL, d_b, d_x, d_p, d_r, d_z, d_temp, d_alpha, d_beta, d_r_norm,
			d_r_norm_old, d_temp_scal, h_x, h_r_norm, M, NNZ, NNL, NNU);

	// allocate memory for the result on host side
	hipDeviceSynchronize();
	// copy result from device to host
	hipMemcpy(h_x, d_x, sizeof(float) * M, hipMemcpyDeviceToHost);

	// compare output with sequential version
	float* h_x_seq = (float *) calloc(M, sizeof(float));
	solveCG_seq(h_A, h_b, h_x_seq, M);

	// for(int i=0; i<M; i++)
	// {
	// 	printf("%2.2f : %2.2f \n",h_x[i], h_x_seq[i]);
	// }
    assert(moreOrLessEqual(h_x, h_x_seq) == 1);

    printf("\nAssertion passed!\n");

	// cleanup memory host
	free(h_A);
	free(h_ACSR);
	free(h_UCSR);
	free(h_LCSR);
	free(h_IA);
	free(h_JA);
	free(h_b);
	free(h_x);
	free(h_r_norm);

	// cleanup memory device
	// hipFree(d_A);
	hipFree(d_ACSR);
	hipFree(d_LD);
	hipFree(d_UD);
	hipFree(d_IA);
	hipFree(d_JA);
	hipFree(d_IL);
	hipFree(d_JL);
	hipFree(d_IU);
	hipFree(d_JU);
	hipFree(d_b);
	hipFree(d_x);
	hipFree(d_p);
	hipFree(d_r);
	hipFree(d_temp);
	hipFree(d_alpha);
	hipFree(d_beta);
	hipFree(d_r_norm);
	hipFree(d_r_norm_old);
	hipFree(d_temp_scal);

	return 0;
}
